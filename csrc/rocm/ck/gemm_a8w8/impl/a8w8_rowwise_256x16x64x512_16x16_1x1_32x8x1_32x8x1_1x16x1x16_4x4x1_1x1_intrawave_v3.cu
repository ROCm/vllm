// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include "../gemm_a8w8_common.cuh"

template <typename DEDataType>
torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y) {
  // The smallest kernel we have available. Works well for memory bound shapes.

  int K = WQ.size(1);
  bool pad = (K % 512 != 0);
  if (pad) {
  using DeviceGemmInstance = DeviceGemmHelper<
      DEDataType,
      256,
      16,
      64,
      512,
      16,
      16,
      1,
      1,
      S<32, 8, 1>,
      S<32, 8, 1>,
      S<1, 16, 1, 16>,
      S<4, 4, 1>,
      1,
      1,
      ck::BlockGemmPipelineScheduler::Intrawave,
      ck::BlockGemmPipelineVersion::v3,
        ck::tensor_operation::device::GemmSpecialization::KPadding>;
  // Run kernel instance.
  return gemm_a8w8_rowwise_impl<DEDataType, DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
  }
  else{
    using DeviceGemmInstance = DeviceGemmHelper<
      DEDataType,
      256,
      16,
      64,
      512,
      16,
      16,
      1,
      1,
      S<32, 8, 1>,
      S<32, 8, 1>,
      S<1, 16, 1, 16>,
      S<4, 4, 1>,
      1,
      1,
      ck::BlockGemmPipelineScheduler::Intrawave,
      ck::BlockGemmPipelineVersion::v3>;
  // Run kernel instance.
  return gemm_a8w8_rowwise_impl<DEDataType, DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
  }
}

template torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3<F16>(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y);

template torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3<B16>(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y);

template <typename DEDataType>
torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3_k4(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y) {
  // The smallest kernel we have available. Works well for memory bound shapes.
  int K = WQ.size(1);
  bool pad = (K % 2048 != 0);
  if (pad) {
  using DeviceGemmInstance = DeviceGemmHelper<
      DEDataType,
      256,
      16,
      64,
      512,
      16,
      16,
      1,
      1,
      S<32, 8, 1>,
      S<32, 8, 1>,
      S<1, 16, 1, 16>,
      S<4, 4, 1>,
      1,
      1,
      ck::BlockGemmPipelineScheduler::Intrawave,
      ck::BlockGemmPipelineVersion::v3,
        ck::tensor_operation::device::GemmSpecialization::KPadding>;
  // Run kernel instance.
  return gemm_a8w8_rowwise_impl<DEDataType, DeviceGemmInstance, 4>(XQ, WQ, x_scale, w_scale, Y);
  }
  else{
    using DeviceGemmInstance = DeviceGemmHelper<
      DEDataType,
      256,
      16,
      64,
      512,
      16,
      16,
      1,
      1,
      S<32, 8, 1>,
      S<32, 8, 1>,
      S<1, 16, 1, 16>,
      S<4, 4, 1>,
      1,
      1,
      ck::BlockGemmPipelineScheduler::Intrawave,
      ck::BlockGemmPipelineVersion::v3>;
  // Run kernel instance.
  return gemm_a8w8_rowwise_impl<DEDataType, DeviceGemmInstance, 4>(XQ, WQ, x_scale, w_scale, Y);
  }
}

template torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3_k4<F16>(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y);

template torch::Tensor
a8w8_rowwise_256x16x64x512_16x16_1x1_32x8x1_32x8x1_1x16x1x16_4x4x1_1x1_intrawave_v3_k4<B16>(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y);
