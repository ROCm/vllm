#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

// declare templates for front (cpp) and back (cuda) sides of function:
// template <typename T>

void LLGemm_Silu(void* in_a, void* in_b, void* out_c, const int M, const int K,
                 hipStream_t stream, const int rows_per_block);
void LLMM_Silu(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
               const int64_t rows_per_block) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  LLGemm_Silu(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
              at::cuda::getCurrentCUDAStream(), rows_per_block);
}

void LLGemm1(void* in_a, void* in_b, void* out_c, const int M, const int K,
             hipStream_t stream, const int rows_per_block);

// template <typename T>
void LLMM1(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
           const int64_t rows_per_block) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  // if (N != in_b.numel())
  //         throw std::invalid_argument("Size mismatch A.numel(): " +
  //         std::to_string(in_a.numel())
  //                           + ", B.numel(): " +
  //                           std::to_string(in_b.numel()));

  // out_c.resize_({N});

  // call the kernel function...
  LLGemm1(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
          at::cuda::getCurrentCUDAStream(), rows_per_block);
}

void wvSpltK_(void* in_a, void* in_b, void* out_c, const int M, const int K,
              const int N, hipStream_t stream, const int CuCount);

void wvSpltK(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
             const int64_t N_in, const int64_t CuCount) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  int N = N_in;
  wvSpltK_(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K, N,
           at::cuda::getCurrentCUDAStream(), CuCount);
}

void wvSpltKQ_(void* in_a, void* in_b, void* out_c, void* scale_a,
               void* scale_b, const int M, const int K, const int Kp,
               const int N, const int Otp_in, hipStream_t stream,
               const int CuCount);

void wvSpltKQ(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
              at::Tensor& scale_a, at::Tensor& scale_b, const int64_t N_in,
              const int64_t Otp_in, const int64_t CuCount) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  auto Kp = in_a.stride(0);
  int N = N_in;
  int Otp = Otp_in;
  wvSpltKQ_(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(),
            scale_a.data_ptr(), scale_b.data_ptr(), M, K, Kp, N, Otp,
            at::cuda::getCurrentCUDAStream(), CuCount);
}

void LLGemmZZ(void* in_a, void* in_b, void* out_c, const int M, const int K,
              hipStream_t stream, const int solidx);

void LLZZ(at::Tensor in_a, at::Tensor in_b, at::Tensor out_c,
          const int64_t solidx = 0) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);

  LLGemmZZ(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
           at::cuda::getCurrentCUDAStream(), solidx);
}
// instantiate the CPP template for T=float:
// template void AddGPU<float>(at::Tensor in_a, at::Tensor in_b, at::Tensor
// out_c);

void MMGPUKernel(float* in_a, float* in_b, float* out_c, int numARows,
                 int numAColumns, int numBRows, int numBColumns, int numCRows,
                 int numCColumns, hipStream_t stream);

void MMCustomGPU(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c) {
  auto matA_sizes{in_a.sizes()};
  auto matB_sizes{in_b.sizes()};
  auto matO_sizes{out_c.sizes()};
  MMGPUKernel(in_a.data_ptr<float>(), in_b.data_ptr<float>(),
              out_c.data_ptr<float>(), matA_sizes[0], matA_sizes[1],
              matB_sizes[0], matB_sizes[1], matO_sizes[0], matO_sizes[1],
              at::cuda::getCurrentCUDAStream());
}
